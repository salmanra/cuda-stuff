#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <string>
#include <hiprand.h>
// #include <thrust/fill.h>

constexpr int TILE_WIDTH = 32; // want this to be geq (?) to block-dim
constexpr int COARSE_FACTOR = 4;
__global__ void cornerTurnMatMul(float *A, float *B, float *C)
{
}

// this is definitely incorrect on the boundary checks.
__global__ void CoarseTiledMatMul(float *A, float *B, float *C, int A_width, int A_height, int B_width)
{
    // One block is responsible for COARSE_FACTOR tiles of output.
    // each thread is responsible for COARSE_FACTOR elts of output.
    // this is in the direction of output columns.
    //
    // "While I have this tile of A loaded, let's handle the computation of many tiles of B"

    // 1. Declare tiles, get row and col indices for this thread.
    // 2. Declare array of COARSE_FACTOR output values.
    // 3. For as many tiles as it takes to cover a row of A
    //     a. Load A into shared memory.
    //     b. for COARSE_FACTOR times:
    //         i. Load B into shared memory. this moves by a tile each iter
    //         ii. matmul loaded tiles. each thread now saves COARSE_FACTOR outputs
    // 4. write all the output values to memory
    //

    // 1.
    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // TILE_WIDTH is equal to block width.
    // each thread is responsible for COARSE_FACTOR consec rows.
    // Thus indexing is as follows:
    int COL_START = COARSE_FACTOR * TILE_WIDTH * blockIdx.x + tx;
    int ROW = TILE_WIDTH * blockIdx.y + ty;

    // has this scheme lost the plot?
    // this array is going straight to DRAM...
    // 2.
    float outVals[COARSE_FACTOR];
    for (int i = 0; i < COARSE_FACTOR; i++)
    {
        outVals[i] = 0.0f;
    }

    // 3.
    for (int i = 0; i < A_width / TILE_WIDTH; i++)
    {
        // a.
        int A_col = i * TILE_WIDTH + tx;
        bool A_idx_valid = (A_col < A_width && ROW < A_height);
        if (A_idx_valid)
            A_tile[ty][tx] = A[ROW * A_width + A_col];

        // b.
        for (int j = 0; j < COARSE_FACTOR; j++)
        {
            // i.
            int B_col = j * TILE_WIDTH + COL_START;
            int B_row = i * TILE_WIDTH + ty;
            bool B_idx_valid = (B_col < B_width && B_row < A_width);
            if (B_idx_valid)
                B_tile[ty][tx] = B[B_row * B_width + B_col];

            __syncthreads(); // don't want to use B_tile before we've written to it

            // ii.
            if (A_idx_valid && B_idx_valid)
            {
                for (int k = 0; k < TILE_WIDTH; k++)
                {
                    outVals[j] += A_tile[ty][k] + B_tile[k][tx];
                }
            }
            __syncthreads(); // don't want to overwrite B_tile while we're still reading B_tile
        }
    }
    for (int i = 0; i < COARSE_FACTOR; i++)
    {
        int C_col = COL_START + i * TILE_WIDTH;
        if (C_col < B_width && ROW < A_height)
            C[ROW * B_width + C_col] = outVals[i];
    }
}

// actually, what is the tiledMatMul impl?
// a 1D grid of blocks that "sweep" over the input arrays?
// a 2D grid of blocks that each matmul a single tile from each input array?
// let this be a square matrix. I wonder does the optim alg for non-square matmul
// tile it into squares anyway? Probably that's the best way to tile.
// assuming tiling is the optimal alg, then you never really do non-square matmul
__global__ void tiledMatMul(float *A, float *B, float *C, int A_height, int A_width, int B_width)
{
    // 1. declare shared mem tiles (one for each input matrix)
    // 2. get row and col of output matrix for this thread
    // 3. For as many tiles as it takes to cover the input matrices (wym by that?)
    //    a. load a single tile elt for each of the two tiles (collaboratively load the tiles)
    //    b. sync on all threads in the block (which is equiv to tile)
    //    c. accum the value of output at row,col into a register
    // 4. store accumulated val into output matrix
    // there is a "redundancy" in how many tiles are loaded to L1 in total
    // each block will end up tiling an entire row of A and col of C.
    // there is some parallelism that we miss out on here methinks
    // because each row of A will get loaded into L1 cache by width/tile_width
    // blocks and same for each col of B. maybe this is the only way, or maybe
    // we can exploit the fact that different blocks actually load in the same
    // tiles as one another. just a thought. part of the premise of cuda is
    // there is no guarantee on synchronicity between blocks. so there could
    // be a way to do it by expanding what a block does (but then it's not so parallel, right?)

    // i'm having an interesting moment looking at the code in the book
    // i think i need to spell out the algorithm in english before
    // internalizing what even in tiledMatMul

    // yo. in 1D blocks, warps are made of threads with consecutive threadIdx.x
    // what about in 2D blocks? "linearized row major layout" -> consecutive threadIdx.x.

    // 1.
    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // output indices of this thread
    // 2.
    int COL = TILE_WIDTH * blockIdx.x + tx;
    int ROW = TILE_WIDTH * blockIdx.y + ty;
    if (ROW >= A_height || COL >= B_width)
    {
        return;
    }
    // let the tile dimensions be the same as the block dimensions.
    //
    float outVal = 0.0;
    // 3.
    for (int i = 0; i < (A_width + TILE_WIDTH - 1) / TILE_WIDTH; i++)
    {
        // a.
        // mem coalescing is done on a warp-level. mem coalescing is a statement about what the hardware does on a warp level
        // accesses to A are clearly coalesced (cont. on tx)
        // accesses to B have the same ty val within a warp (assuming tile-width >= 32)
        // ergo accesses to B are coalesced (cont. on COL)
        int A_col = i * TILE_WIDTH + tx;
        int B_row = i * TILE_WIDTH + ty;
        if (A_col >= A_width || B_row >= A_height)
        {
            return;
        }
        A_tile[ty][tx] = A[ROW * A_width + A_col]; // row is const, col within a tile is const, col by tile is not const
        B_tile[ty][tx] = B[B_row * B_width + COL];
        // b.
        __syncthreads();
        // now matmul the tiles
        // c.
        for (int j = 0; j < TILE_WIDTH; j++)
        {
            outVal += A_tile[ty][j] * B_tile[j][tx];
        }
        __syncthreads();
    }
    // 4.
    C[ROW * B_width + COL] = outVal;
}

//  row matmul sucks!!! from the perspective of a single thread, access to A is
// contiguous over time. But that means that on a single iteration, each thread in
// the block is accessing a different row of A, which is a totally non-cont
// access pattern! That means that B is a contiguous access pattern tho,
// and maybe that's more important? Yes, because at least one load will have
// to be performed on be on each iteratin of the inner loop, but
// you only have to load from A on each iter of the outer loop.
// So if mem access is slow once every outer loop, but fast on
// every inner loop, that's preferable. Which means row matmul is
// preferable as written to colmatmul.
__global__ void rowMatmul(float *A, float *B, float *C, int i, int j, int k)
{
    int row = blockDim.x + blockIdx.x + threadIdx.x;

    if (row >= i)
    {
        return;
    }

    for (int x = 0; x < k; x++)
    {
        float val = 0.0f;
        for (int y = 0; y < j; y++)
        {
            // smart is to get this row of A into some registers
            // can't (it's an array)! put it into L1 cache via shared memory
            val += A[row * j + y] + B[y * k + x];
        }
        C[row * k + x] = val;
    }
}

__global__ void colMatmul(float *A, float *B, float *C, int i, int j, int k)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col >= k)
    {
        return;
    }

    for (int z = 0; z < i; z++)
    {
        float val = 0.0f;
        for (int y = 0; y < j; y++)
        {
            val += A[z * j + y] * B[y * k + col];
        }
        C[z * i + col] = val;
    }
}

__global__ void matmul(float *A, float *B, float *C, int i, int j, int k)
{
    // is each thread handling a single elt of the output matrix?

    // let's say yes, and that the output matrix is ixk
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row >= i || col >= k)
    {
        return;
    }
    float val = 0.0f;
    for (int idx = 0; idx < j; idx++)
    {
        // use registers!!! no other thread accesses C at this index. we can localize this.
        val += A[row * j + idx] * B[idx * k + col];
        // ^^ this part stays in memory, ^^ and ^^ get loaded in as needed
        // what if
        //                      ^^ this stays (bc row major?)
        // and ^^               and             ^^ get loaded in as needed?
    }
    C[row * k + col] = val;
}

void naiveMatmul(float *A, float *B, float *C, int i, int j, int k)
{

    // "row" and "col" are row and col of output.
    // input shapes are A -> ixj, B->jxk
    for (int row = 0; row < i; row++)
    {
        for (int col = 0; col < k; col++)
        {
            float tmp = 0.0f;
            for (int y = 0; y < j; y++)
            {
                // j is the shared dim,
                // the dim we are collapsing,
                // so it goes in the inner

                // of note: j to index in A is to jump from row to row
                // (there are j elts in a row of A)
                // k fills this role for B (there are k elts in a row of B)
                tmp += A[row * j + y] * B[y * k + col];
            }
            C[row * k + col] = tmp;
        }
    }
}

void initMat(float *mat, int h, int w)
{
    for (int i = 0; i < w; i++)
    {
        for (int j = 0; j < h; j++)
        {
            // mat[j * w + i] = (float)j;
            // mat[j * w + i] = (float)(j * w + i) / (i * j + 1);
            mat[j * w + i] = 3.0f;
        }
    }
}

int main()
{
    // today I want to test tiled, coarsened, non-sqaure matmul
    int i, j, k;
    float *a, *b, *c, *d_a, *d_b, *d_c;

    // do some stuff with the constants
    i = 200;
    j = 200;
    k = 200;

    a = (float *)malloc(i * j * sizeof(float));
    b = (float *)malloc(j * k * sizeof(float));
    c = (float *)malloc(i * k * sizeof(float));

    hipMalloc(&d_a, i * j * sizeof(float));
    hipMalloc(&d_b, j * k * sizeof(float));
    hipMalloc(&d_c, i * k * sizeof(float));

    initMat(a, i, j);
    initMat(b, j, k);

    // a thing to remember about memset is that it fills as many bytes with the given value,
    // not as many "array elts" (it's not an array, just a pointer to some bytes!).
    // thus setting to 0 is easy because float(0) is four bytes each with the value 0
    memset((void *)c, 0, i * k * sizeof(float));

    hipMemcpy(d_a, a, i * j * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, j * k * sizeof(float), hipMemcpyHostToDevice);

    // TODO: investigate thrust::fill. I do want to learn how to memset in cuda, but for now let's just copy c over
    // hipMemcpy(d_c, c, i * k * sizeof(float), hipMemcpyHostToDevice);

    int blockLength = TILE_WIDTH;
    dim3 threadsPerBlock(blockLength, blockLength);
    dim3 gridShape((k + blockLength - 1) / blockLength, (i + blockLength - 1) / blockLength);

    std::cout << "grid x: " << gridShape.x << " grid y: " << gridShape.y << std::endl;
    std::cout << "block x: " << threadsPerBlock.x << " blok y: " << threadsPerBlock.y << std::endl;

    tiledMatMul<<<gridShape, threadsPerBlock>>>(d_a, d_b, d_c, i, j, k);
    // matmul<<<gridShape, threadsPerBlock>>>(d_a, d_b, d_c, i, j, k);
    // CoarseTiledMatMul<<<gridShape, threadsPerBlock>>>(d_a, d_b, d_c, i, j, k);

    hipDeviceSynchronize();
    // verify C
    // to veryify correctness, I guess we can serially calculate A@B and compare
    naiveMatmul(a, b, c, i, j, k);

    float *res;
    res = (float *)malloc(i * k * sizeof(float));
    hipMemcpy(res, d_c, i * k * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    int rescount = 0;
    int ccount = 0;

    for (int u = 0; u < i; u++)
    {
        for (int w = 0; w < k; w++)
        {
            maxError = max(maxError, abs(res[u * k + w] - c[u * k + w]));
            if (abs(res[u * k + w] - c[u * k + w]) > 0.0000001)
            {
                rescount++;
                // std::cout << u << ' ' << w << std::endl;
            }
            // std::cout << res[u * k + w] << ' ' << c[u * k + w] << std::endl;
            // if (abs(c[u * i + w] - 360.0f) > 0.00001)
            // {
            //     ccount++;
            // }
        }
        // std::cout << res[u] << std::endl;
        // std::cout << c[u] << std::endl;
    }
    std::cout << "Max error: " << maxError << std::endl;
    std::cout << "Bad Res: " << rescount << std::endl;
    // std::cout << "Bad c: " << ccount << std::endl;
    // std::cout << res[0] << std::endl;
    // std::cout << c[0] << std::endl;

    free(a);
    free(b);
    free(c);
    free(res);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}