#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) // bounds check. doesn't this affect warp eligibility?
        y[i] = a * x[i] + y[i];
}

int main()
{
    int N = 1 << 20;
    float *x, *y, *d_x, *d_y;
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0;
        y[i] = 2.0;
    }
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = N + blockSize - 1;
    saxpy<<<numBlocks, blockSize>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = max(maxError, abs(y[i] - 4.0f));
    }
    printf("Max Error: %f\n", maxError);
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}